
#include <hip/hip_runtime.h>
template <typename T>  __global__  void kernelgpuInitu(T *f, T *xdg, T *uinf, T *param, int modelnumber, int ng, int ncx, int nce, int npe, int ne)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		int j = i%npe;
		int k = (i-j)/npe;
		T xdg1 = xdg[j+npe*0+npe*ncx*k];
		T xdg2 = xdg[j+npe*1+npe*ncx*k];
		T t2 = xdg1*xdg1;
		T t3 = xdg2*xdg2;
		f[j+npe*0+npe*nce*k] = exp(t2*-5.0E+3-t3*5.0E+3);
		f[j+npe*1+npe*nce*k] = exp(t2*(-2.958579881656805E+3)-t3*2.958579881656805E+3);
		f[j+npe*2+npe*nce*k] = exp(t2*(-1.020408163265306E+4)-t3*1.020408163265306E+4);
		f[j+npe*3+npe*nce*k] = 0.0;
		i += blockDim.x * gridDim.x;
	}
}

template <typename T> void gpuInitu(T *f, T *xdg, T *uinf, T *param, int modelnumber, int ng, int ncx, int nce, int npe, int ne)
{
	int blockDim = 256;
	int gridDim = (ng + blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuInitu<<<gridDim, blockDim>>>(f, xdg, uinf, param, modelnumber, ng, ncx, nce, npe, ne);
}

template void gpuInitu(double *, double *, double *, double *, int, int, int, int, int, int);
template void gpuInitu(float *, float *, float *, float *, int, int, int, int, int, int);